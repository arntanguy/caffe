#include "hip/hip_runtime.h"
// Copyright 2013 Yangqing Jia

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {

template <typename Dtype>
Dtype SiameseLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  return Forward_cpu(bottom, top);
}

template <typename Dtype>
void SiameseLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_CLASS(SiameseLossLayer);


}  // namespace caffe
